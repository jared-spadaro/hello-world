#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/features2d/features2d.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/video/video.hpp>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>

#include "tracker.hpp"

#define MIN_FEATURES 30
#define FAST_THRESHOLD 50
#define FEATURE_POINT_SIZE 3
#define MAX_FRAMES 30
#define MASK_RADIUS 3
#define NUM_PIX 16
#define NUM_CONTIG 12
#define THREAD_SIZE 16

using namespace cv;
using namespace std;

__device__ int d_numCorners;

__global__ void fastDetector(uchar *d_ImGreyVec, uchar *d_contigCounts, 
 int thresh, int minContigCorner, int numRows, int numCols) {
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;

    int row = by*blockDim.y + ty;
    int col = bx*blockDim.x + tx;
    
    if (((row < (numRows - MASK_RADIUS)) && (row >= MASK_RADIUS)) &&
        ((col < (numCols - MASK_RADIUS)) && (col >= MASK_RADIUS))) {
        
        uchar cirPixels[16];
        int ndx = row * numCols + col;
        
        // variables to keep track the number of contiguous pixels of same label
        int contiguous = 0;
        int begContig = 0;
        int maxContig = 0;
        int beginning;

        // variable to hold result of comparison
        int compareResult;

        // Each thread examines pixel at Row y, Col x
        uchar centerPix = d_ImGreyVec[ndx];
        int maxThresh = centerPix + thresh;
        int minThresh = centerPix - thresh;

        /*
           Get pixel values along circle that are used to check if center
           pixel is a corner 
        */
        cirPixels[0] = d_ImGreyVec[ndx - 3];
        cirPixels[1] = d_ImGreyVec[ndx - numCols - 3];
        cirPixels[2] = d_ImGreyVec[ndx - (2 * numCols) - 2];
        cirPixels[3] = d_ImGreyVec[ndx - (3 * numCols) - 1];
        cirPixels[4] = d_ImGreyVec[ndx - (3 * numCols)];
        cirPixels[5] = d_ImGreyVec[ndx - (3 * numCols) + 1];
        cirPixels[6] = d_ImGreyVec[ndx - (2 * numCols) + 2];
        cirPixels[7] = d_ImGreyVec[ndx - (numCols) + 3];
        cirPixels[8] = d_ImGreyVec[ndx + 3];
        cirPixels[9] = d_ImGreyVec[ndx + (numCols) + 3];
        cirPixels[10] = d_ImGreyVec[ndx + (2 * numCols) + 2];
        cirPixels[11] = d_ImGreyVec[ndx + (3 * numCols) + 1];
        cirPixels[12] = d_ImGreyVec[ndx + (3 * numCols)];
        cirPixels[13] = d_ImGreyVec[ndx + (3 * numCols) - 1];
        cirPixels[14] = d_ImGreyVec[ndx + (2 * numCols) - 2];
        cirPixels[15] = d_ImGreyVec[ndx + (1 * numCols) - 3];

        for (int i = 0; i < NUM_PIX; i++) {
            compareResult = cirPixels[i] > maxThresh;
            contiguous = contiguous * compareResult + compareResult;
            if (beginning && contiguous == 0) {
                beginning = 0;
            }
            if (beginning) {
                begContig++;
            }
            if (contiguous > maxContig) {
                maxContig = contiguous;
            }
        }
        contiguous += begContig;
        if (maxContig < contiguous) {
            maxContig = contiguous;
        }

        for (int i = 0; i < NUM_PIX; i++) {
            compareResult = cirPixels[i] < minThresh;
            contiguous = contiguous * compareResult + compareResult;
            if (beginning && contiguous == 0) {
                beginning = 0;
            }
            if (beginning) {
                begContig++;
            }
            if (contiguous > maxContig) {
                maxContig = contiguous;
            }
        }
        contiguous += begContig;
        if (maxContig < contiguous) {
            maxContig = contiguous;
        }
        if (maxContig >= minContigCorner) {
            d_contigCounts[ndx] = maxContig;
        }
    }

}

int main(int argc, char** argv )
{
  /* Video Camera Object */
  VideoCapture cam;

  /* Video Frame Images */
  Mat frame, frame_gray, oldframe;
  uchar *imGreyPtr;

  /* Create vectors and/or pointers that reference host/device data */
  thrust::host_vector<uchar> h_ImGreyVec;
  thrust::device_vector<uchar> d_ImGreyVec;
  uchar *d_contigCounts;
  uchar *h_contigCounts;
  
  int numRows, numCols, size;

  /* Matrix for Rigid Transform */
  Mat rigid_transform;

  /* Feature Point Vectors */
  std::vector<KeyPoint> keypoints;
  std::vector<Point2f> pts, tracked_pts, prev_pts, saved_pts;

  /* Status and Error Flags for KLT Algorithm */
  std::vector<uchar> status;
  std::vector<float> err;

  /* Pixed Distance Calculation Variables */
  float dist_x = 0, dist_y = 0;

  /* Frame Counter */
  int frame_count = 0;

  /* Open Camera Device */
  cam.open("vid.mov");

  /* Check Camera Device Opens (Returns -1 on Error) */
  if (!cam.isOpened()) {
     return -1;
  }

  /* Create Window for Displaying Video Feed */
  namedWindow("Feature Tracking", WINDOW_NORMAL);

  cam >> frame;
  numRows = frame.rows;
  numCols = frame.cols;
  size = numRows*numCols;
  int goodSize = size;
  h_contigCounts = (uchar *)malloc(size * sizeof(uchar)); 
  /* initialize cuda kernel parameters */
  int gridSizeX = ((numRows-1) / THREAD_SIZE) + 1;
  int gridSizeY = ((numCols-1) / THREAD_SIZE) + 1;
  dim3 blockSize(THREAD_SIZE, THREAD_SIZE, 1);
  dim3 gridSize(gridSizeX, gridSizeY, 1);
  hipMalloc((void **) &d_contigCounts, size * sizeof(uchar)); 
  uchar *d_ImGrey_raw;
  int error;

  /* Loop Forever Until Camera Closed */
  while (numRows) {
     /* Convert Frame to Grayscale */
     cvtColor(frame, frame_gray, COLOR_BGR2GRAY);
     imGreyPtr = (uchar *)frame_gray.ptr<uchar>(0);
     h_ImGreyVec.assign(imGreyPtr, imGreyPtr + size);
     d_ImGreyVec = h_ImGreyVec;
     d_ImGrey_raw = thrust::raw_pointer_cast(&d_ImGreyVec[0]);
     fastDetector<<<gridSize, blockSize>>>(d_ImGrey_raw,        
      d_contigCounts, FAST_THRESHOLD, NUM_CONTIG, numRows, numCols);
     error = hipDeviceSynchronize();
     /*imshow("Live View", frame);
     if (waitKey(1) >= 0) {
        break;
     }*/

     /* Obtain Frame from Camera */
     cam >> frame;
     numRows = frame.rows;
     numCols = frame.cols;
     size = numRows*numCols;
     frame_count++;
  }

  cout << "made it" << endl;
  hipMemcpy(h_contigCounts, d_contigCounts, size * sizeof(uchar),
   hipMemcpyDeviceToHost);
  for (int i = 0; i < 10; i++) {
    cout << h_contigCounts[i] << " ";
  }
  cout << "\n";
  hipFree(d_contigCounts);
  
  return 0;
}
