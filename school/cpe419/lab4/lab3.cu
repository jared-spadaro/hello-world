#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : OpenCVCUDA.cu
 Author      : 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

/*
 ============================================================================
 Name        : OpenCVCu.cu
 Author      :
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
#include <opencv2/opencv.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <iostream>
#include <numeric>
#include <stdlib.h>


cv::Mat imageRGBA;
cv::Mat imageGrey;
uchar4        *d_rgbaImage__;

unsigned char *d_greyImage__;

size_t numRows() { return imageRGBA.rows; }
size_t numCols() { return imageRGBA.cols; }

//returns a pointer to an RGBA version of the input image
//and a pointer to the single channel grey-scale output
//on both the host and device

//Histogram: allocates host and device arrays for histogram
void preProcess(uchar4 **inputImage, unsigned char **greyImage,
                uchar4 **d_rgbaImage, unsigned char **d_greyImage,
                unsigned char **histogram, unsigned char **d_histogram,
                const std::string &filename) {
	//make sure the context initializes ok
	hipFree(0);
	cv::Mat image;
	image = cv::imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
	if (image.empty()) {
		std::cerr << "Couldn't open file: " << filename << std::endl;
		exit(1);
	}

	cv::cvtColor(image, imageRGBA, CV_BGR2RGBA);

	//allocate memory for the output
	imageGrey.create(image.rows, image.cols, CV_8UC1);
	//This shouldn't ever happen given the way the images are created
	//at least based upon my limited understanding of OpenCV, but better to check
	if (!imageRGBA.isContinuous() || !imageGrey.isContinuous()) {
		std::cerr << "Images aren't continuous!! Exiting." << std::endl;
		exit(1);
	}
	*inputImage = (uchar4 *)imageRGBA.ptr<unsigned char>(0);
	*greyImage  = imageGrey.ptr<unsigned char>(0);
	const size_t numPixels = numRows() * numCols();
	//allocate memory on the device for both input and output
	hipMalloc(d_rgbaImage, sizeof(uchar4) * numPixels);
	hipMalloc(d_greyImage, sizeof(unsigned char) * numPixels);
	hipMemset(*d_greyImage, 0, numPixels * sizeof(unsigned char)); //make sure no memory is left laying around

        //Histogram allocation. We need arrays of size numPixels 
        *histogram = (unsigned char *)calloc(sizeof(unsigned char), 256);
        hipMalloc(d_histogram, sizeof(unsigned char) * 256); 
        hipMemset(*d_histogram, 0, sizeof(unsigned char) * numPixels);

	//copy input array to the GPU
	hipMemcpy(*d_rgbaImage, *inputImage, sizeof(uchar4) * numPixels,hipMemcpyHostToDevice);
	d_rgbaImage__ = *d_rgbaImage;
	d_greyImage__ = *d_greyImage;
}

void postProcess(const std::string& output_file, unsigned char *eq_image) {
	const int numPixels = numRows() * numCols();
	//copy the output back to the host
	hipMemcpy(d_greyImage__, eq_image, sizeof(unsigned char) * numPixels, hipMemcpyHostToDevice); 
        hipMemcpy(imageGrey.ptr<unsigned char>(0), d_greyImage__, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost);
        //output the image
	cv::imwrite(output_file.c_str(), imageGrey);
	cv::imshow ("Output Image", imageGrey);
	cv::waitKey(0);
	//cleanup
	hipFree(d_rgbaImage__);
	hipFree(d_greyImage__);
}

void referenceCalculation(const uchar4* const rgbaImage,
                          unsigned char *const greyImage,
                          size_t numRows,
                          size_t numCols)
{
	for (size_t r = 0; r < numRows; ++r) {
		for (size_t c = 0; c < numCols; ++c) {
			uchar4 rgba = rgbaImage[r * numCols + c];
			float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
			greyImage[r * numCols + c] = channelSum;
		}
	}
}

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
	//Fill in the kernel to convert from color to greyscale
	//the mapping from components of a uchar4 to RGBA is:
	// .x -> R ; .y -> G ; .z -> B ; .w -> A
	//
	//The output (greyImage) at each pixel should be the result of
	//applying the formula: output = .299f * R + .587f * G + .114f * B;
	//Note: We will be ignoring the alpha channel for this conversion
	//First create a mapping from the 2D block and grid locations
	//to an absolute 2D location in the image, then use that to
	//calculate a 1D offset.

	int row= blockIdx.y * blockDim.y + threadIdx.y; // row Number
	int col = blockIdx.x * blockDim.x + threadIdx.x; // col Number
	if( row < numRows || col < numCols ){
			int i = row*numCols+col;
			uchar4 rgba = rgbaImage[ i ];
			float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
			greyImage[ i ] = channelSum;
	}
}
void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage,
                            uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage,
                            size_t numRows,
                            size_t numCols)
{
	int threadSize=32;
	int gridSizeX=((numRows-1)/threadSize)+1;
	int gridSizeY=((numCols-1)/threadSize)+1;
	const dim3 blockSize(threadSize, threadSize, 1);  //TODO
	const dim3 gridSize(gridSizeY, gridSizeX, 1);  //TODO
	rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage,d_greyImage,numRows,numCols);
	hipDeviceSynchronize(); hipGetLastError();
}

__global__ void histogram_kernel (unsigned char *d_histogram, unsigned char *d_greyImage, 
 int numRows, int numCols) {
   int row= blockIdx.y * blockDim.y + threadIdx.y; // row Number
   int col = blockIdx.x * blockDim.x + threadIdx.x; // col Number
   if( row < numRows || col < numCols ){
      int i = row*numCols+col;
      //atomicAdd((int *)(d_histogram + d_greyImage[i]), 1);  
      	d_histogram[d_greyImage[i]]++;
   }
}

void normalizeHistogram(unsigned char *histogram) {
   int i;
   int *c_histogram = (int *)calloc(sizeof(int),  256);
   int running_total = 0;
   for (i = 0; i < 256; i++) {
      running_total += histogram[i];
      c_histogram[i] = running_total;
   }
   for (i = 0; i < 256; i++) {
      printf("%d ", c_histogram[i]);
   }
   printf("\n\n");
   //now normalize
   for (i = 0; i < 256; i++) {
      histogram[i] = ((c_histogram[i] - 1) * 255) / (numRows() * numCols());
   }
   for (i = 0; i < 256; i++) {
      printf("%d ", histogram[i]);
   }
   printf("\n");
}
   

void createHistogram(unsigned char *histogram, unsigned char *d_histogram, unsigned char *d_greyImage,
 size_t numRows, size_t numCols) {
   int threadSize = 32, ndx = 0, numPixels = 0;
   int gridSizeX = ((numRows-1)/threadSize)+1;
   int gridSizeY = ((numCols-1)/threadSize)+1;
   const dim3 blockSize(threadSize, threadSize, 1);
   const dim3 gridSize(gridSizeY, gridSizeX, 1);
   histogram_kernel<<<gridSize, blockSize>>>(d_histogram, d_greyImage, numRows, numCols);
   hipDeviceSynchronize(); hipGetLastError();
   hipMemcpy(histogram, d_histogram, sizeof(unsigned char) * 256, hipMemcpyDeviceToHost);
   for(ndx = 0; ndx < 256; ndx++) {
      printf("%d ", histogram[ndx]);
      numPixels += histogram[ndx];
   }
   printf("\n");
   printf("%d\n", numPixels);
   printf("\n");
   normalizeHistogram(histogram);
}

int main(int argc, char **argv) {
	uchar4        *h_rgbaImage, *d_rgbaImage;
	unsigned char *h_greyImage, *d_greyImage;
        unsigned char *histogram, *d_histogram, *eq_image;
        int i;
	std::string input_file;
	std::string output_file;
	if (argc == 3) {
		input_file  = std::string(argv[1]);
		output_file = std::string(argv[2]);
	}
	else {
		std::cerr << "Usage: ./hw input_file output_file" << std::endl;
		exit(1);
	}
	//load the image and give us our input and output pointers
	preProcess(&h_rgbaImage, &h_greyImage, &d_rgbaImage, &d_greyImage, 
         &histogram, &d_histogram, input_file);
	//call the students' code
	your_rgba_to_greyscale(h_rgbaImage, d_rgbaImage, d_greyImage, numRows(), numCols());
        //h_greyImage = (unsigned char *)calloc(numPixels, sizeof(unsigned char));
	const size_t numPixels = numRows() * numCols();
        eq_image = (unsigned char *)calloc(numPixels, sizeof(unsigned char));
	hipMemcpy(h_greyImage, d_greyImage__, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost);
        //BUILD HISTOGRAM HERE (write kernel, call it here)
        createHistogram(histogram, d_histogram, d_greyImage, numRows(), numCols());
        printf("sample histogram value = %d\n", histogram[27]);
        //for (i = 0; i < numPixels; i++) {
          // printf("%d ", h_greyImage[i]);
        //} 
        printf("\n");
        for (i = 0; i < numPixels; i++) {
           eq_image[i] = histogram[h_greyImage[i]];
           printf("%d ", eq_image[i]);
        }
        //printf("\n");
	hipDeviceSynchronize();
	hipGetLastError();
	printf("\n");
	//check results and output the grey image
	postProcess(output_file, eq_image);
	return 0;
}

