#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : ImageBlur.cu
 Author      : 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <opencv2/opencv.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include <numeric>
#include <stdlib.h>
//This code does Convolution on an array
//it has thw two implementations 2D and 2 pass 1D convolution
//the filter is just a [1 1 1 1 1]
//you should change it to your desired filte
#define BLUR_SIZE 5
#define TILE_WIDTH 16
__constant__ float M_d[BLUR_SIZE];

__global__ void convolution_1D_basic(float *P, float *N,int width){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  float PValue=0;
  int N_start_point=i-(BLUR_SIZE/2);
  for(int j=0; j<BLUR_SIZE; j++){
	  if (N_start_point+j>=0 && N_start_point+j < width) {
		  PValue += N[N_start_point+j]*M_d[j];
	  }
  }
  //printf("thread %d %lf \n",i,PValue);
  P[i]=PValue/BLUR_SIZE;
}

__global__ void blurKernel(float *out, float *in, int width, int height) {

  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (col < width && row < height) {
    float pixVal = 0; //when change into read an image this should be an integer
    int pixels = 0;

    // Get the average of the surrounding BLUR_SIZE x BLUR_SIZE box
    for (int blurrow = -BLUR_SIZE; blurrow < BLUR_SIZE + 1; ++blurrow) {
      for (int blurcol = -BLUR_SIZE; blurcol < BLUR_SIZE + 1; ++blurcol) {
    	int currow = row + blurrow;
        int curcol = col + blurcol;
        // Verify we have a valid image pixel
        if (currow > -1 && currow < height && curcol > -1 && curcol < width) {
          pixVal += in[currow * width + curcol];
          pixels++; // Keep track of number of pixels in the avg
         }
      }
    }
    // Write our new pixel value out
    //out[row * width + col] = (unsigned char)(pixVal / pixels); // when changed into a image input do this
    out[row * width + col] = (pixVal / pixels);
  }
}

int main(int argc, char *argv[]) {


	unsigned int imageWidth=64;
	unsigned int imageHeight=64;
	dim3 dimsA(imageHeight,imageHeight);
	unsigned int size_A=dimsA.x*dimsA.y;
	unsigned int mem_size_A=sizeof(float)*size_A;


	//allocate mem for the host data
	float *hostInputImageData=(float *)malloc(mem_size_A);
	float *hostOutputImageData=(float *)malloc(mem_size_A);;

	//init array in host //in future this will be reading from an image
	for(int i=0; i<size_A;i++){
		hostInputImageData[i]=0.5f;
		hostOutputImageData[i]=0.0f;
	}
	//@@@ Debug code  print input image just to make sure we a reading non garbage
	//for(int i=1; i<size_A;i++)
	//		printf("input %i = %lf\n", i,hostOutputImageData[i]);


	//allocate mem for GPU Data
	float *deviceInputImageData;
	float *deviceOutputImageData;
	hipMalloc((void **)&deviceInputImageData, mem_size_A);
	hipMalloc((void **)&deviceOutputImageData, mem_size_A);

	//copy from the CPU memory to the GPU Memory
	hipMemcpy(deviceInputImageData, hostInputImageData,mem_size_A,hipMemcpyHostToDevice);

	//@@@@ Doing computation on the GPU 2D Image blur
	//dim3 dimGrid(ceil((float)imageWidth / TILE_WIDTH),ceil((float)imageHeight / TILE_WIDTH));
	//dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	//@@@ Debug print dimGrid and block
	//printf("dimBlock=%d dimGrid=%d", dimBlock.x, dimGrid.x);
	//blurKernel<<<dimGrid, dimBlock>>>(deviceOutputImageData, deviceInputImageData, dimsA.x,dimsA.y);

	//@@@ Doing computation on the GPU for the 1D Image blur rows
	float M_h[5]={1,1,1,1,1};
	hipMemcpyToSymbol(HIP_SYMBOL(M_d),M_h, BLUR_SIZE*sizeof(float));
	int dimGrid=(ceil((float)size_A / TILE_WIDTH));
	int dimBlock=TILE_WIDTH;
	printf("dimBlock=%d dimGrid=%d", dimBlock, dimGrid);
	//rows
	convolution_1D_basic<<<dimGrid, dimBlock>>>(deviceOutputImageData, deviceInputImageData, size_A);
	hipDeviceSynchronize();hipGetLastError();
	//@@@ Debug code print new array blurred
	//for(int i=1; i<size_A;i++)
	//	printf("\n%lf",hostOutputImageData[i]);
	//cols
	float *deviceOutputImageDataCol;
	hipMalloc((void **)&deviceOutputImageDataCol, mem_size_A);

	convolution_1D_basic<<<dimGrid, dimBlock>>>(deviceOutputImageDataCol, deviceOutputImageData, size_A);
	hipDeviceSynchronize();hipGetLastError();

	//done with computation on GPU so copy back to CPU
	hipMemcpy(hostOutputImageData, deviceOutputImageData,imageWidth * imageHeight * sizeof(float),hipMemcpyDeviceToHost);

	//free memory from cuda
	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);

	//@@@ Debug code print new array blurred
	for(int i=1; i<size_A;i++)
		printf("\n%lf",hostOutputImageData[i]);
	//free CPU memory
	delete(hostInputImageData);
	delete(hostOutputImageData);
	return 0;


}
