#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <pthread.h>

#define ROWS1 10
#define COLS1 10
#define ROWS2 10
#define COLS2 10

#define NUMTHREADS 16

/* global variables shared by all functions/threads */
double **h_vals1, **h_vals2;
double **d_vals1, **d_vals2;
double **h_product;
double **d_product;

void *Multiply(void *);

void InitializeDeviceMem () {
   int i;
   hipMalloc((void ***)&d_vals1, sizeof(double *) * ROWS1);
   hipMalloc((void ***)&d_vals2, sizeof(double *) * ROWS2);
   hipMalloc((void ***)&d_product, sizeof(double *) * ROWS1);
   
   /* initialize data structures */
   for (i = 0; i < ROWS1; i++) {
      hipMalloc((void **)(d_vals1 + i), sizeof(double) * COLS1);
      hipMalloc((void **)(d_product + i), sizeof(double) * COLS2);
   }
   for (i = 0; i < ROWS2; i++) {
      hipMalloc((void **)(d_vals2 + i), sizeof(double) * COLS2);
   }

   hipMemcpy(d_vals1, h_vals1, xxxYYY);
}


/* read matrix files, multiply them, and print result to file */
void Read () {
   int i, j;

   FILE *stream1 = fopen("matrix1", "r");
   FILE *stream2 = fopen("matrix2", "r");
   FILE *result = fopen("result", "w");
   
   /* data structures for holding matrices and result */
   h_vals1 = (double **)malloc(sizeof(double *) * ROWS1);
   h_vals2 = (double **)malloc(sizeof(double *) * ROWS2);
   h_product = (double **)malloc(sizeof(double *) * ROWS1);
   
   /* initialize data structures */
   for (i = 0; i < ROWS1; i++) {
      *(h_vals1 + i) = (double *)malloc(sizeof(double) * ROWS1);
      *(h_product + i) = (double *)malloc(sizeof(double) * ROWS2);
   }
   for (i = 0; i < ROWS2; i++) {
      *(h_vals2 + i) = (double *)malloc(sizeof(double) * COLS2);
   }
   
   if (COLS1 != ROWS2) {
      fprintf(stderr, "Dimensions of matrices do not allow for multiplication\n");
      exit(1);
   }

   /* read */
   for (i = 0; i < ROWS1; i++) {
      for (j = 0; j < COLS1; j++) {
         fscanf(stream1, "%lf", *(h_vals1 + i) + j);
      }
   }
   for (i = 0; i < ROWS2; i++) {
      for (j = 0; j < COLS2; j++) {
         fscanf(stream2, "%lf", *(h_vals2 + i) + j);
      }
   }
}

/* thread function. each thread will perform multiplication on some number of rows.
 * timing of the multiplication occurs here.
 */
__global__ void Multiply(double **d_vals1, double **d_vals2, double **d_product, int width) {
   int i;
   //int range, first, last;
   //struct timespec begin, end;
   //int thread_id = (int)id;   
   //double elapsed;
   
   //clock_gettime(CLOCK_THREAD_CPUTIME_ID, &begin);
   
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   int col = blockIdx.x * blockDim.x + threadIdx.x;

   double result = 0;

   for (i = 0; i < width; i++) {
      result += d_vals1[row][i] * d_vals2[i][col];
   }
   
   d_product[row][col] = result;

   //clock_gettime(CLOCK_THREAD_CPUTIME_ID, &end);
   //elapsed = end.tv_sec - begin.tv_sec;
   //elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;
  
   /* print time elapsed */
   //printf("Time = %lf\n", elapsed);
}

int main (int argc, char **argv) {
   //double time = 0;
   Read(); /* host memory is allocated and initialized here */
   InitializeDeviceMem(); /* device memory is allocated here */
   Multiply<<<(COLS1/NUMTHREADS, COLS1/NUMTHREADS), (NUMTHREADS, NUMTHREADS)>>>
    (d_vals1, d_vals2, d_product, COLS1);   
   int i, j;
   for (i = 0; i < ROWS1; i++) {
      for (j = 0; j < COLS2; j++) {
         printf("%lf", d_product[i][j]);
      }
   }
   return 0;
}
